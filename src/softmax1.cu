#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__ void softmax(float *x, int N)
{
    // Index for each thread
    int index = threadIdx.x;
    float max_val = 0.0f;
    float sum_exp = 0.0f;

    // Find maximum value from input array
    for (int i = 0; i < N; i++)
    {
        if (x[i] > max_val)
            max_val = x[i];
    }

    // Calculate the sum of exponentials
    for (int i = 0; i < N; i++)
    {
        sum_exp += expf(x[i] - max_val);
    }

    // Finally, apply the softmax function for the particular thread
    x[index] = expf(x[index] - max_val) / sum_exp;
}

// Main function
int main()
{
    int N = 5;
    float h_x[N] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    float *d_x;

    // Allocate the memory on the GPU
    hipMalloc(&d_x, N*sizeof(float));

    // Copy the array 'h_x' to the GPU
    hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);

    // Call the Kernel function
    softmax<<<1, N>>>(d_x, N);

    // Copy back the result array to the CPU
    hipMemcpy(h_x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

    // Free the memory
    hipFree(d_x);
    printf("Softmax output:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%.6f ", h_x[i]);
    }
    printf("\n");

    return 0;
}
